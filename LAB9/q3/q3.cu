
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mul1(int *A, int *B, int *C, int n, int q){
	int id = threadIdx.x, i,j;
	for(i=0;i<q;i++){
		C[id*q+i] = 0;
		for(j=0;j<n;j++)
			C[id*q+i] += A[id*n+j] * B[j*q+i];
	}
}

__global__ void mul2(int *A, int *B, int *C, int m, int q){
	int id = threadIdx.x, i, j, n = blockDim.x;
	for(i=0;i<m;i++){
		C[i*q+id] = 0;
		for(j=0;j<n;j++)
			C[i*q+id] += A[i*n+j] * B[j*q+id];
	}
}

__global__ void mul3(int *A, int *B, int *C, int n){
	int bid = blockIdx.x, tid = threadIdx.x, q = blockDim.x,i;
	C[bid*q+tid] = 0;
	for(i=0;i<n;i++)
		C[bid*q+tid] += A[bid*n+i] * B[i*q+tid];
}

int main(){
	int *a,*b,*c,*da,*db,*dc,m,n,p,q,i,j;

	printf("Enter m: ");
	scanf("%d",&m);
	printf("Enter n: ");
	scanf("%d",&n);

	int size1 = sizeof(int)*m*n;

	a = (int *)malloc(size1);

	printf("Enter first matrix:\n");
	for(i=0;i<m*n;i++)
		scanf("%d",&a[i]);


	printf("Enter p: ");
	scanf("%d",&p);
	printf("Enter q: ");
	scanf("%d",&q);

	int size2 = sizeof(int)*p*q;

	b = (int *)malloc(size2);

	printf("Enter second matrix:\n");
	for(i=0;i<p*q;i++)
		scanf("%d",&b[i]);

	if(n!=p){
		printf("%d != %d. Cannot multiply.\n",n,p);
		exit(0);
	}

	int size3 = sizeof(int)*m*q;
	c = (int *)malloc(size3);

	hipMalloc((void **)&da,size1);
	hipMalloc((void **)&db,size2);
	hipMalloc((void **)&dc,size3);

	hipMemcpy(da,a,size1,hipMemcpyHostToDevice);
	hipMemcpy(db,b,size2,hipMemcpyHostToDevice);

	printf("Result 1)Thread per row:\n");

	mul1<<<1,m>>>(da,db,dc,n,q);

	hipMemcpy(c,dc,size3,hipMemcpyDeviceToHost);

	for(i=0;i<m;i++){
		for(j=0;j<q;j++)
			printf("%d ",c[i*q+j]);
		printf("\n");
	}

	printf("Result 2)Thread per column:\n");

	mul2<<<1,n>>>(da,db,dc,m,q);

	hipMemcpy(c,dc,size3,hipMemcpyDeviceToHost);

	for(i=0;i<m;i++){
		for(j=0;j<q;j++)
			printf("%d ",c[i*q+j]);
		printf("\n");
	}

	printf("Result 3)Thread per element:\n");

	mul3<<<m,q>>>(da,db,dc,n);

	hipMemcpy(c,dc,size3,hipMemcpyDeviceToHost);

	for(i=0;i<m;i++){
		for(j=0;j<q;j++)
			printf("%d ",c[i*q+j]);
		printf("\n");
	}

	hipFree(da);
	hipFree(db);
	hipFree(dc);
	return 0;


}
