
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void complement(int *A, int *B){
	int i = blockIdx.x, j = threadIdx.x, m = gridDim.x, n = blockDim.x,k=1,temp,t=0,rev;
	if(i!=0 && i!=m-1 && j!=0 && j!=n-1){
		temp = A[i*n+j];
		do{
			t = t*10 + !(temp%2);
			temp /= 2;
		}while(temp>0);
		do{
			temp = t%10;
			rev = rev*10 + temp;
			t /= 10;
		}while(t>0);
		B[i*n+j] = rev;
	}
	else
		B[i*n+j] = A[i*n+j];
}

int main(){
	int *a,*t,m,n,i,j,*da,*dt;
	printf("Enter m: ");
	scanf("%d",&m);
	printf("Enter n: ");
	scanf("%d",&n);

	int size = sizeof(int)*m*n;

	a = (int *)malloc(size);
	t = (int *)malloc(size);

	printf("Enter the matrix:\n");
	for(i=0;i<m*n;i++)
		scanf("%d",&a[i]);

	hipMalloc((void **)&da,size);
	hipMalloc((void **)&dt,size);

	hipMemcpy(da,a,size,hipMemcpyHostToDevice);

	complement<<<m,n>>>(da,dt);

	hipMemcpy(t,dt,size,hipMemcpyDeviceToHost);

	printf("Result:\n");

	for(i=0;i<m;i++){
		for(j=0;j<n;j++)
			printf("%d ",t[i*n+j]);
		printf("\n");
	}

	hipFree(da);
	hipFree(dt);
	return 0;
}

