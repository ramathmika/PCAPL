
#include <hip/hip_runtime.h>
#include<stdio.h>
#include <stdlib.h>

__global__ void replaceMat(int *mat, int *res){
	int i = blockIdx.x, j = threadIdx.x, n = blockDim.x;
	//res[i*n+j] = mat[i*n+j];
	res[i*n+j] = powf(mat[i*n+j],i+1);
	//for(int p=0;p<i;p++)
		//res[i*n+j] *= mat[i*n+j];
}

int main(){
	int *a,*t,m,n,i,j,*da,*dt;
	printf("Enter m: ");
	scanf("%d",&m);
	printf("Enter n: ");
	scanf("%d",&n);

	int size = sizeof(int)*m*n;

	a = (int *)malloc(size);
	t = (int *)malloc(size);

	printf("Enter the matrix:\n");
	for(i=0;i<m*n;i++)
		scanf("%d",&a[i]);

	hipMalloc((void **)&da,size);
	hipMalloc((void **)&dt,size);

	hipMemcpy(da,a,size,hipMemcpyHostToDevice);

	replaceMat<<<m,n>>>(da,dt);

	hipMemcpy(t,dt,size,hipMemcpyDeviceToHost);

	printf("Result:\n");

	for(i=0;i<m;i++){
		for(j=0;j<n;j++)
			printf("%d ",t[i*n+j]);
		printf("\n");
	}

	hipFree(da);
	hipFree(dt);
	return 0;
}
