
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add1(int *A, int *B, int *C, int n){
	int id = threadIdx.x, i;
	for(i=0;i<n;i++){
		C[id*n+i] = A[id*n+i] + B[id*n+i];
	}
}

__global__ void add2(int *A, int *B, int *C, int m){
	int id = threadIdx.x, i, n = blockDim.x;
	for(i=0;i<m;i++)
		C[i*n+id] = A[i*n+id] + B[i*n+id];
}

__global__ void add3(int *A, int *B, int *C){
	int i = blockIdx.x, j = threadIdx.x, n = blockDim.x;
	C[i*n+j] = A[i*n+j] + B[i*n+j];
}

int main(){
	int *a,*b,*c,*da,*db,*dc,m,n,i,j;

	printf("Enter m: ");
	scanf("%d",&m);
	printf("Enter n: ");
	scanf("%d",&n);

	int size = sizeof(int)*m*n;

	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);

	printf("Enter first matrix:\n");
	for(i=0;i<m*n;i++)
		scanf("%d",&a[i]);

	printf("Enter second matrix:\n");
	for(i=0;i<m*n;i++)
		scanf("%d",&b[i]);

	hipMalloc((void **)&da,size);
	hipMalloc((void **)&db,size);
	hipMalloc((void **)&dc,size);

	hipMemcpy(da,a,size,hipMemcpyHostToDevice);
	hipMemcpy(db,b,size,hipMemcpyHostToDevice);

	printf("Result 1)Thread per row:\n");

	add1<<<1,m>>>(da,db,dc,n);

	hipMemcpy(c,dc,size,hipMemcpyDeviceToHost);

	for(i=0;i<m;i++){
		for(j=0;j<n;j++)
			printf("%d ",c[i*n+j]);
		printf("\n");
	}

	printf("Result 2)Thread per column:\n");

	add2<<<1,n>>>(da,db,dc,m);

	hipMemcpy(c,dc,size,hipMemcpyDeviceToHost);

	for(i=0;i<m;i++){
		for(j=0;j<n;j++)
			printf("%d ",c[i*n+j]);
		printf("\n");
	}

	printf("Result 3)Thread per element:\n");

	add3<<<m,n>>>(da,db,dc);

	hipMemcpy(c,dc,size,hipMemcpyDeviceToHost);

	for(i=0;i<m;i++){
		for(j=0;j<n;j++)
			printf("%d ",c[i*n+j]);
		printf("\n");
	}


	hipFree(da);
	hipFree(db);
	hipFree(dc);
	return 0;


}
