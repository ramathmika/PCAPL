#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void transpose(int *a, int *t){
	int n = threadIdx.x, m = blockIdx.x, size = blockDim.x, size1 = gridDim.x;
	t[n*size1+m] = a[m*size+n];
}

int main(){
	int *a,*t,m,n,i,j,*da,*dt;
	printf("Enter m: ");
	scanf("%d",&m);
	printf("Enter n: ");
	scanf("%d",&n);

	int size = sizeof(int)*m*n;

	a = (int *)malloc(size);
	t = (int *)malloc(size);

	printf("Enter the matrix:\n");
	for(i=0;i<m*n;i++)
		scanf("%d",&a[i]);

	hipMalloc((void **)&da,size);
	hipMalloc((void **)&dt,size);

	hipMemcpy(da,a,size,hipMemcpyHostToDevice);

	transpose<<<m,n>>>(da,dt);

	hipMemcpy(t,dt,size,hipMemcpyDeviceToHost);

	printf("Result:\n");

	for(i=0;i<n;i++){
		for(j=0;j<m;j++)
			printf("%d ",t[i*m+j]);
		printf("\n");
	}

	hipFree(da);
	hipFree(dt);
	return 0;
}
