
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matrixSort(int *A, int *B){
	int id = blockIdx.x*blockDim.x + threadIdx.x,m = gridDim.x,pos=0;
	for(int i=0;i<m;i++)
		if(A[id]>A[i] || (A[i]==A[id] && i<id))
			pos++;
	B[pos] = A[id];

}


int main(){
	int A[100][100],B[100],m,n,i,j,size;

	printf("Enter value for n,m: ");
	scanf("%d %d",&n,&m);

	printf("Enter the values for matrix :\n");
	for(i=0;i<n;i++)
		for(j=0;j<m;j++)
			scanf("%d",&A[i][j]);


	int *da,*db;
	size = sizeof(int)*n*m;

	hipMalloc((void **)&da,size);
	hipMalloc((void**)&db,size/n);

	hipMemcpy(da,A,size,hipMemcpyHostToDevice);


	printf("Result:\n");
	for(i=0;i<n;i++){
		matrixSort<<<n,m>>>(da,db);

		hipMemcpy(B,db,size,hipMemcpyDeviceToHost);
		for(j=0;j<m;j++)
			printf("%d ",B[i]);
		printf("\n");
	}



	hipFree(da);
	hipFree(db);
	return 0;
}
