
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vecAdd(int *X, int *Y, int a, int *F){
	int id = threadIdx.x;
	F[id] = a*X[id] + Y[id];
}

int main(){
	int i,n,a,X[100],Y[100],F[100],*dx,*dy,*df;

	printf("Enter value for a: ");
		scanf("%d",&a);

	printf("Enter value for n: ");
	scanf("%d",&n);

	printf("Enter the values for vector X:\n");
	for(i=0;i<n;i++)
		scanf("%d",&X[i]);
	printf("Enter the values for vector Y:\n");
	for(i=0;i<n;i++)
		scanf("%d",&Y[i]);

	int size = sizeof(int)*n;

	hipMalloc((void **)&dx,size);
	hipMalloc((void **)&dy,size);
	hipMalloc((void **)&df,size);

	hipMemcpy(dx,X,size,hipMemcpyHostToDevice);
	hipMemcpy(dy,Y,size,hipMemcpyHostToDevice);

	vecAdd<<<1,n>>>(dx,dy,a,df);

	hipMemcpy(F,df,size,hipMemcpyDeviceToHost);

	printf("Result: ");
	for(i=0;i<n;i++)
		printf("%d ",F[i]);

	hipFree(dx);
	hipFree(dy);
	hipFree(df);

	return 0;
}
