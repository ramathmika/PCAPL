
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vecAdd1(int *A, int *B,int *C){
	int id = blockIdx.x;
	C[id] = A[id] + B[id];
}

__global__ void vecAdd2(int *A, int *B, int *C){
	int id = threadIdx.x;
	C[id] = A[id] + B[id];
}

__global__ void vecAdd3(int *A, int *B, int *C){
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	C[id] = A[id] + B[id];
}

int main(){
	int A[100],B[100],C[100],n,i,size;

	printf("Enter value for n: ");
	scanf("%d",&n);

	printf("Enter the values for vector A:\n");
	for(i=0;i<n;i++)
		scanf("%d",&A[i]);
	printf("Enter the values for vector B:\n");
	for(i=0;i<n;i++)
		scanf("%d",&B[i]);

	int *da, *db, *dc;
	size = sizeof(int)*n;

	hipMalloc((void **)&da,size);
	hipMalloc((void**)&db,size);
	hipMalloc((void **)&dc,size);

	hipMemcpy(da,A,size,hipMemcpyHostToDevice);
	hipMemcpy(db,B,size,hipMemcpyHostToDevice);


	printf("Result through %d blocks:\n",n);
	vecAdd1<<<n,1>>>(da,db,dc);

	hipMemcpy(C,dc,size,hipMemcpyDeviceToHost);

	for(i=0;i<n;i++)
		printf("%d ",C[i]);

	printf("\nResult through %d threads:\n",n);
	vecAdd2<<<1,n>>>(da,db,dc);

	hipMemcpy(C,dc,size,hipMemcpyDeviceToHost);

	for(i=0;i<n;i++)
		printf("%d ",C[i]);

	printf("\nResult through varying block size:\n");
	vecAdd3<<<ceil(n/3),3>>>(da,db,dc);

	hipMemcpy(C,dc,size,hipMemcpyDeviceToHost);

	for(i=0;i<n;i++)
		printf("%d ",C[i]);

	hipFree(da);
	hipFree(db);
	hipFree(dc);
	return 0;
}
