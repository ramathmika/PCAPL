
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vecSine(float *A, float *B){
	int id = blockIdx.x;
	B[id] = sin(A[id]);
}


int main(){
	float A[100],B[100];
	int n,i,size;

	printf("Enter value for n: ");
	scanf("%d",&n);

	printf("Enter the values for vector A in radians :\n");
	for(i=0;i<n;i++)
		scanf("%f",&A[i]);


	float *da,*db;
	size = sizeof(float)*n;

	hipMalloc((void **)&da,size);
	hipMalloc((void**)&db,size);

	hipMemcpy(da,A,size,hipMemcpyHostToDevice);


	printf("Result: ");
	vecSine<<<n,1>>>(da,db);

	hipMemcpy(B,db,size,hipMemcpyDeviceToHost);

	for(i=0;i<n;i++)
		printf("%f ",B[i]);


	hipFree(da);
	hipFree(db);
	return 0;
}
